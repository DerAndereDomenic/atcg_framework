#include "hip/hip_runtime.h"
//#define EIGEN_NO_CUDA
#include <Registration/CPDBackendCUDA.h>
#include <cutil.h>

namespace atcg
{
    namespace detail
    {
        double Pmn(const Eigen::Vector3d& x, const Eigen::Vector3d& y, double var)
        {
            return std::exp(-0.5f/var*(x-y).dot(x-y));
        }

        __global__ void fillP(double* X, double* Y, double* P, double* R, double* t, double* Z, double s, double var, uint32_t N, uint32_t M)
        {
            const size_t tid = cutil::globalThreadIndex();

            if(tid >= M*N)
                return;

            auto [n,m] = cutil::index1Dto2D(tid, N);

            double* x = X + 3*n;
            double* y = Y + 3*m;

            double d[3];

            d[0] = s*(R[0 + 0*3] * y[0] + R[0 + 1*3] * y[1] + R[0 + 2*3] * y[2]) + t[0] - x[0];
            d[1] = s*(R[1 + 0*3] * y[0] + R[1 + 1*3] * y[1] + R[1 + 2*3] * y[2]) + t[1] - x[1];
            d[2] = s*(R[2 + 0*3] * y[0] + R[2 + 1*3] * y[1] + R[2 + 2*3] * y[2]) + t[2] - x[2];

            P[tid] = std::exp(-0.5f/var * (d[0]*d[0] + d[1]*d[1] + d[2]*d[2]));

            atomicAdd(&Z[n], P[tid]);
        }

        __device__ double atomicMul(double* address, double val)
        {
            unsigned long long int* address_as_ull =
                                    (unsigned long long int*)address;
            unsigned long long int old = *address_as_ull, assumed;

            do {
                assumed = old;
                old = atomicCAS(address_as_ull, assumed,
                                __double_as_longlong(val *
                                    __longlong_as_double(assumed)));

            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);

            return __longlong_as_double(old);
        }

        __global__ void normalize(double* P, double* PX, double* PY, double* Z, uint32_t N, uint32_t M)
        {
            const size_t tid = cutil::globalThreadIndex();

            if(tid >= N*M)
            {
                return;
            }

            auto [n,m] = cutil::index1Dto2D(tid, N);

            //TODO: Normalize P
            atomicMul(&P[tid], 1.0/Z[n]);
            atomicAdd(&PX[n], P[tid]);
            atomicAdd(&PY[m], P[tid]);
        }
    }

    class CPDBackendCUDA::Impl
    {
    public:

        Impl() = default;

        ~Impl();

        double* devX,* devY,* devP,* devZ;
        double* devR, * devT;
        double* devPX,* devPY;
        uint32_t N,M;
    };

    CPDBackendCUDA::Impl::~Impl()
    {
        cudaSafeCall(hipFree(devX));
        cudaSafeCall(hipFree(devY));
        cudaSafeCall(hipFree(devP));
        cudaSafeCall(hipFree(devR));
        cudaSafeCall(hipFree(devT));
        cudaSafeCall(hipFree(devZ));
        cudaSafeCall(hipFree(devPX));
        cudaSafeCall(hipFree(devPY));
    }
    
    CPDBackendCUDA::CPDBackendCUDA( RowMatrix& X,  RowMatrix& Y)
        :CPDBackend(X,Y)
    {
        impl = std::make_unique<Impl>();
        impl->N = X.rows();
        impl->M = Y.rows();
        cudaSafeCall(hipMalloc((void**)&(impl->devX), sizeof(double) * impl->N * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devY), sizeof(double) * impl->M * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devP), sizeof(double) * impl->M * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devZ), sizeof(double) * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devPX), sizeof(double) * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devPY), sizeof(double) * impl->M));
        

        cudaSafeCall(hipMalloc((void**)&(impl->devR), sizeof(double) * 3 * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devT), sizeof(double) * 3));

        cudaSafeCall(hipMemcpy((void*)(impl->devX), (void*)&X(0), sizeof(double) * impl->N * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy((void*)(impl->devY), (void*)&Y(0), sizeof(double) * impl->M * 3, hipMemcpyHostToDevice));
    }

    void CPDBackendCUDA::estimate(const Transformation& transform,
                                 RowMatrix& P, 
                                 Eigen::VectorXd& PX, 
                                 Eigen::VectorXd& PY, 
                                 double bias, 
                                 double var)
    {
        cudaSafeCall(hipMemcpy((void*)(impl->devR), (void*)&transform.R(0), sizeof(double) * 3 * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy((void*)(impl->devT), (void*)&transform.t(0), sizeof(double) * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemset((void*)impl->devZ, bias, sizeof(double) * impl->N));
        cudaSafeCall(hipMemset((void*)impl->devPX, 0, sizeof(double) * impl->N));
        cudaSafeCall(hipMemset((void*)impl->devPY, 0, sizeof(double) * impl->M));

        cutil::KernelSize config = cutil::configureKernel(impl->N * impl->M);
        detail::fillP<<<config.blocks, config.threads>>>(impl->devX,
                                                         impl->devY,
                                                         impl->devP,
                                                         impl->devR,
                                                         impl->devT,
                                                         impl->devZ,
                                                         transform.s,
                                                         var,
                                                         impl->N,
                                                         impl->M);
        cutil::syncStream();

        detail::normalize<<<config.blocks, config.threads>>>(impl->devP,
                                                             impl->devPX,
                                                             impl->devPY,
                                                             impl->devZ,
                                                             impl->N,
                                                             impl->M);
        cutil::syncStream();

        cudaSafeCall(hipMemcpy((void*)&P(0), (void*)(impl->devP), sizeof(double) * impl->N * impl->M, hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy((void*)&PX(0), (void*)(impl->devPX), sizeof(double) * impl->N, hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy((void*)&PY(0), (void*)(impl->devPY), sizeof(double) * impl->M, hipMemcpyDeviceToHost));

        //Eigen::VectorXd Z = Eigen::VectorXd::Zero(impl->N);
        //cudaSafeCall(hipMemcpy((void*)&Z(0), (void*)(impl->devZ), sizeof(double) * impl->N, hipMemcpyDeviceToHost));
        /*for(size_t m = 0; m < impl->M; ++m)
        {
            for(size_t n = 0; n < impl->N; ++n)
            {
                Z(n) += P(m,n);
            }
        }*/

        /*for(size_t m = 0; m < impl->M; ++m)
        {
            for(size_t n = 0; n < impl->N; ++n)
            {
                P(m,n) = P(m,n)/Z(n);
                PX(n) += P(m,n); //PT1
                PY(m) += P(m,n); //P1
            }
        }*/
    }
}