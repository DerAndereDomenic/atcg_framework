#include "hip/hip_runtime.h"
#define EIGEN_NO_CUDA
#include <Registration/CPDBackendCUDA.h>

#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>

#include <DataStructure/Timer.h>
#include <DataStructure/Statistics.h>

namespace atcg
{
    namespace detail
    {
        struct Pmn
        {
            double* X;
            double* Y;
            double* R;
            double* t;
            double s;
            double var;
            uint32_t n;

            Pmn(double* X, double* Y, double* R, double* t, double s, double var, uint32_t n)
                :X(X),Y(Y),R(R),t(t),s(s),var(var), n(n)
            {}

            __device__
            inline double operator()(uint32_t m)
            {
                double* x = X + 3*n;
                double* y = Y + 3*m;

                double d[3];

                d[0] = s*(R[0 + 0*3] * y[0] + R[0 + 1*3] * y[1] + R[0 + 2*3] * y[2]) + t[0] - x[0];
                d[1] = s*(R[1 + 0*3] * y[0] + R[1 + 1*3] * y[1] + R[1 + 2*3] * y[2]) + t[1] - x[1];
                d[2] = s*(R[2 + 0*3] * y[0] + R[2 + 1*3] * y[1] + R[2 + 2*3] * y[2]) + t[2] - x[2];

                return std::exp(-0.5f/var * (d[0]*d[0] + d[1]*d[1] + d[2]*d[2]));
            }
        };

        __global__ void fillP(double* X, double* Y, double* P, double* R, double* t, double* Z, double s, double var, uint32_t N, uint32_t M)
        {
            const size_t tid = cutil::globalThreadIndex();

            if(tid >= M*N)
                return;

            auto [n,m] = cutil::index1Dto2D(tid, N);

            double* x = X + 3*n;
            double* y = Y + 3*m;

            double d[3];

            d[0] = s*(R[0 + 0*3] * y[0] + R[0 + 1*3] * y[1] + R[0 + 2*3] * y[2]) + t[0] - x[0];
            d[1] = s*(R[1 + 0*3] * y[0] + R[1 + 1*3] * y[1] + R[1 + 2*3] * y[2]) + t[1] - x[1];
            d[2] = s*(R[2 + 0*3] * y[0] + R[2 + 1*3] * y[1] + R[2 + 2*3] * y[2]) + t[2] - x[2];

            P[tid] = std::exp(-0.5f/var * (d[0]*d[0] + d[1]*d[1] + d[2]*d[2]));

            atomicAdd(&Z[n], P[tid]);
        }

        __global__ void normalize(double* P, double* PX, double* PY, double* Z, double* Np, uint32_t N, uint32_t M)
        {
            const size_t tid = cutil::globalThreadIndex();

            if(tid >= N*M)
            {
                return;
            }

            auto [n,m] = cutil::index1Dto2D(tid, N);

            P[tid] = P[tid]/(Z[n] + 1e-12);
            atomicAdd(&PX[n], P[tid]);
            atomicAdd(&PY[m], P[tid]);
            atomicAdd(Np, P[tid]);
        }
    }

    class CPDBackendCUDA::Impl
    {
    public:

        Impl() = default;

        ~Impl();

        double* devX,* devY,* devP,* devZ;
        double* devR, * devT;
        double* devPX,* devPY;
        double* devNp;
        uint32_t N,M;

        RowMatrix P;
    };

    CPDBackendCUDA::Impl::~Impl()
    {
        cudaSafeCall(hipFree(devX));
        cudaSafeCall(hipFree(devY));
        cudaSafeCall(hipFree(devP));
        cudaSafeCall(hipFree(devR));
        cudaSafeCall(hipFree(devT));
        cudaSafeCall(hipFree(devZ));
        cudaSafeCall(hipFree(devPX));
        cudaSafeCall(hipFree(devPY));
        cudaSafeCall(hipFree(devNp));
    }
    
    CPDBackendCUDA::CPDBackendCUDA( RowMatrix& X,  RowMatrix& Y)
        :CPDBackend(X,Y)
    {
        impl = std::make_unique<Impl>();
        impl->N = X.rows();
        impl->M = Y.rows();
        impl->P = RowMatrix::Zero(impl->M, impl->N);
        cudaSafeCall(hipMalloc((void**)&(impl->devX), sizeof(double) * impl->N * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devY), sizeof(double) * impl->M * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devP), sizeof(double) * impl->M * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devZ), sizeof(double) * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devPX), sizeof(double) * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devPY), sizeof(double) * impl->M));
        cudaSafeCall(hipMalloc((void**)&(impl->devNp), sizeof(double)));
        

        cudaSafeCall(hipMalloc((void**)&(impl->devR), sizeof(double) * 3 * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devT), sizeof(double) * 3));

        cudaSafeCall(hipMemcpy((void*)(impl->devX), (void*)&X(0), sizeof(double) * impl->N * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy((void*)(impl->devY), (void*)&Y(0), sizeof(double) * impl->M * 3, hipMemcpyHostToDevice));
    }

    void CPDBackendCUDA::estimate(const Transformation& transform,
                                 Eigen::VectorXd& PX, 
                                 Eigen::VectorXd& PY, 
                                 double& Np,
                                 double bias, 
                                 double var)
    {
        cudaSafeCall(hipMemcpy((void*)(impl->devR), (void*)&transform.R(0), sizeof(double) * 3 * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy((void*)(impl->devT), (void*)&transform.t(0), sizeof(double) * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemset((void*)impl->devZ, bias, sizeof(double) * impl->N));
        cudaSafeCall(hipMemset((void*)impl->devPX, 0, sizeof(double) * impl->N));
        cudaSafeCall(hipMemset((void*)impl->devPY, 0, sizeof(double) * impl->M));
        cudaSafeCall(hipMemset((void*)impl->devNp, 0, sizeof(double)));

        cutil::KernelSize config = cutil::configureKernel(impl->N * impl->M);
        detail::fillP<<<config.blocks, config.threads>>>(impl->devX,
                                                         impl->devY,
                                                         impl->devP,
                                                         impl->devR,
                                                         impl->devT,
                                                         impl->devZ,
                                                         transform.s,
                                                         var,
                                                         impl->N,
                                                         impl->M);
        cutil::syncStream();

        detail::normalize<<<config.blocks, config.threads>>>(impl->devP,
                                                             impl->devPX,
                                                             impl->devPY,
                                                             impl->devZ,
                                                             impl->devNp,
                                                             impl->N,
                                                             impl->M);
        cutil::syncStream();

        cudaSafeCall(hipMemcpy((void*)&impl->P(0), (void*)(impl->devP), sizeof(double) * impl->N * impl->M, hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy((void*)&PX(0), (void*)(impl->devPX), sizeof(double) * impl->N, hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy((void*)&PY(0), (void*)(impl->devPY), sizeof(double) * impl->M, hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy((void*)&Np, (void*)(impl->devNp), sizeof(double), hipMemcpyDeviceToHost));

        Np = 1.0/Np;

        /*Eigen::VectorXd Z(impl->N);
        Statistic<float> stats("thrust");
        for(uint32_t n = 0; n < impl->N; ++n)
        {
            Timer t;
            Z[n] = thrust::transform_reduce(thrust::device,
                                            thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(impl->M),
                                            detail::Pmn(impl->devX, impl->devY, impl->devR, impl->devT, transform.s, var, n),
                                            0,
                                            thrust::plus());
            stats.addSample(t.elapsedMillis());
        }
        std::cout << stats;*/
        //for(m -> M)
        //double result = thrust::reduce(impl->devP, impl->devP + impl->N*impl->M, 0, thrust::plus());
        //int m = 0;
        //thrust::transform_reduce(thrust::device, thrust::counting_iterator<int>(0), thrust::constant_iterator<int>(impl->N), [&](int n){return impl->devP[m + n*impl->N];}, 0, thrust::plus());
    }

    void CPDBackendCUDA::maximize(const RowMatrix& XC,
                                  const RowMatrix& YC,
                                  RowMatrix& A)
    {
        A = XC.transpose() * impl->P.transpose() * YC;
    }
}