#include "hip/hip_runtime.h"
//#define EIGEN_NO_CUDA
#include <Registration/CPDBackendCUDA.h>
#include <cutil.h>

namespace atcg
{
    namespace detail
    {
        double Pmn(const Eigen::Vector3d& x, const Eigen::Vector3d& y, double var)
        {
            return std::exp(-0.5f/var*(x-y).dot(x-y));
        }

        __global__ void fillP(double* X, double* Y, double* P, double* R, double* t, double* Z, double s, double var, uint32_t N, uint32_t M)
        {
            const size_t tid = cutil::globalThreadIndex();

            if(tid >= M*N)
                return;

            auto [n,m] = cutil::index1Dto2D(tid, N);

            double* x = X + 3*n;
            double* y = Y + 3*m;

            double d[3];

            d[0] = s*(R[0 + 0*3] * y[0] + R[0 + 1*3] * y[1] + R[0 + 2*3] * y[2]) + t[0] - x[0];
            d[1] = s*(R[1 + 0*3] * y[0] + R[1 + 1*3] * y[1] + R[1 + 2*3] * y[2]) + t[1] - x[1];
            d[2] = s*(R[2 + 0*3] * y[0] + R[2 + 1*3] * y[1] + R[2 + 2*3] * y[2]) + t[2] - x[2];

            P[tid] = std::exp(-0.5f/var * (d[0]*d[0] + d[1]*d[1] + d[2]*d[2]));

            atomicAdd(&Z[n], P[tid]);
        }
    }

    class CPDBackendCUDA::Impl
    {
    public:

        Impl() = default;

        ~Impl();

        double* devX,* devY,* devP,* devZ;
        double* devR, * devT;
        uint32_t N,M;
    };

    CPDBackendCUDA::Impl::~Impl()
    {
        cudaSafeCall(hipFree(devX));
        cudaSafeCall(hipFree(devY));
        cudaSafeCall(hipFree(devP));
        cudaSafeCall(hipFree(devR));
        cudaSafeCall(hipFree(devT));
        cudaSafeCall(hipFree(devZ));
    }
    
    CPDBackendCUDA::CPDBackendCUDA( RowMatrix& X,  RowMatrix& Y)
        :CPDBackend(X,Y)
    {
        impl = std::make_unique<Impl>();
        impl->N = X.rows();
        impl->M = Y.rows();
        cudaSafeCall(hipMalloc((void**)&(impl->devX), sizeof(double) * impl->N * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devY), sizeof(double) * impl->M * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devP), sizeof(double) * impl->M * impl->N));
        cudaSafeCall(hipMalloc((void**)&(impl->devZ), sizeof(double) * impl->N));
        

        cudaSafeCall(hipMalloc((void**)&(impl->devR), sizeof(double) * 3 * 3));
        cudaSafeCall(hipMalloc((void**)&(impl->devT), sizeof(double) * 3));

        cudaSafeCall(hipMemcpy((void*)(impl->devX), (void*)&X(0), sizeof(double) * impl->N * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy((void*)(impl->devY), (void*)&Y(0), sizeof(double) * impl->M * 3, hipMemcpyHostToDevice));
    }

    void CPDBackendCUDA::estimate(const Transformation& transform,
                                 RowMatrix& P, 
                                 Eigen::VectorXd& PX, 
                                 Eigen::VectorXd& PY, 
                                 double bias, 
                                 double var)
    {
        cudaSafeCall(hipMemcpy((void*)(impl->devR), (void*)&transform.R(0), sizeof(double) * 3 * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy((void*)(impl->devT), (void*)&transform.t(0), sizeof(double) * 3, hipMemcpyHostToDevice));
        cudaSafeCall(hipMemset((void*)impl->devZ, bias, sizeof(double) * impl->N));

        cutil::KernelSize config = cutil::configureKernel(impl->N * impl->M);
        detail::fillP<<<config.blocks, config.threads>>>(impl->devX,
                                                         impl->devY,
                                                         impl->devP,
                                                         impl->devR,
                                                         impl->devT,
                                                         impl->devZ,
                                                         transform.s,
                                                         var,
                                                         impl->N,
                                                         impl->M);
        cutil::syncStream();

        cudaSafeCall(hipMemcpy((void*)&P(0), (void*)(impl->devP), sizeof(double) * impl->N * impl->M, hipMemcpyDeviceToHost));

        Eigen::VectorXd Z = Eigen::VectorXd::Zero(impl->N);
        cudaSafeCall(hipMemcpy((void*)&Z(0), (void*)(impl->devZ), sizeof(double) * impl->N, hipMemcpyDeviceToHost));
        /*for(size_t m = 0; m < impl->M; ++m)
        {
            for(size_t n = 0; n < impl->N; ++n)
            {
                Z(n) += P(m,n);
            }
        }*/

        for(size_t m = 0; m < impl->M; ++m)
        {
            for(size_t n = 0; n < impl->N; ++n)
            {
                P(m,n) = P(m,n)/Z(n);
                PX(n) += P(m,n); //PT1
                PY(m) += P(m,n); //P1
            }
        }
    }
}