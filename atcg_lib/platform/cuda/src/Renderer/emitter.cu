#include "hip/hip_runtime.h"
#include <Renderer/Emitter.h>
#include <Renderer/Common.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <c10/cuda/CUDAGuard.h>

namespace atcg
{

namespace detail
{
__global__ void
computeMeshTrianglePDFKernel(const torch::PackedTensorAccessor32<float, 2, at::RestrictPtrTraits> positions,
                             const torch::PackedTensorAccessor32<int, 2, at::RestrictPtrTraits> indices,
                             const glm::mat4 transform,
                             torch::PackedTensorAccessor32<float, 1, at::RestrictPtrTraits> pdf)
{
    auto id = static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x) + static_cast<int64_t>(threadIdx.x);
    auto num_threads = static_cast<int64_t>(gridDim.x) * static_cast<int64_t>(blockDim.x);
    for(auto tid = id; tid < indices.size(0); tid += num_threads)
    {
        if(tid >= indices.size(0)) return;

        glm::u32vec3 triangle_indices = glm::u32vec3(indices[tid][0], indices[tid][1], indices[tid][2]);
        glm::vec3 local_P0            = glm::vec3(positions[triangle_indices.x][0],
                                       positions[triangle_indices.x][1],
                                       positions[triangle_indices.x][2]);
        glm::vec3 local_P1            = glm::vec3(positions[triangle_indices.y][0],
                                       positions[triangle_indices.y][1],
                                       positions[triangle_indices.y][2]);
        glm::vec3 local_P2            = glm::vec3(positions[triangle_indices.z][0],
                                       positions[triangle_indices.z][1],
                                       positions[triangle_indices.z][2]);

        glm::vec3 P0 = glm::vec3(transform * glm::vec4(local_P0, 1));
        glm::vec3 P1 = glm::vec3(transform * glm::vec4(local_P1, 1));
        glm::vec3 P2 = glm::vec3(transform * glm::vec4(local_P2, 1));

        // Compute triangle area
        float parallelogram_area = glm::length(glm::cross(P1 - P0, P2 - P0));
        float triangle_area      = 0.5f * parallelogram_area;

        // Write unnormalized pdf
        pdf[tid] = triangle_area;
    }
}

__global__ void computeMeshTriangleCDFKernel(torch::PackedTensorAccessor32<float, 1, at::RestrictPtrTraits> cdf)
{
    float acc = 0;
    for(uint32_t i = 0; i < cdf.size(0); ++i)
    {
        acc += cdf[i];
        cdf[i] = acc;
    }
}

__global__ void normalizeMeshTriangleCDFKernel(torch::PackedTensorAccessor32<float, 1, at::RestrictPtrTraits> cdf,
                                               float total_value)
{
    auto id = static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x) + static_cast<int64_t>(threadIdx.x);
    auto num_threads = static_cast<int64_t>(gridDim.x) * static_cast<int64_t>(blockDim.x);
    for(auto tid = id; tid < cdf.size(0); tid += num_threads)
    {
        if(tid >= cdf.size(0)) return;

        cdf[tid] /= total_value;
    }
}
}    // namespace detail

MeshEmitter::MeshEmitter(const atcg::ref_ptr<Graph>& graph, const glm::mat4& transform, const Material& material)
{
    auto emissive_texture = material.getEmissiveTexture()->getData(atcg::GPU);

    MeshEmitterData data;

    ::detail::convertToTextureObject(emissive_texture, _emissive_texture, data.emissive_texture);

    data.emitter_scaling = material.emission_scale;

    _positions = graph->getPositions(atcg::GPU).clone();
    _normals   = graph->getNormals(atcg::GPU).clone();
    _uvs       = graph->getUVs(atcg::GPU).clone();
    _faces     = graph->getFaces(atcg::GPU).clone();

    data.positions = (glm::vec3*)_positions.data_ptr();
    data.normals   = (glm::vec3*)_normals.data_ptr();
    data.uvs       = (glm::vec3*)_uvs.data_ptr();
    data.faces     = (glm::u32vec3*)_faces.data_ptr();

    _mesh_cdf = torch::zeros({_faces.size(0)}, atcg::TensorOptions::floatDeviceOptions());

    auto device = _mesh_cdf.device();

    {
        at::cuda::CUDAGuard device_guard {device};
        const auto stream = at::cuda::getCurrentCUDAStream();

        const int threads_per_block = 128;
        dim3 grid;
        at::cuda::getApplyGrid(_faces.size(0), grid, device.index(), threads_per_block);
        dim3 threads = at::cuda::getApplyBlock(threads_per_block);

        detail::computeMeshTrianglePDFKernel<<<grid, threads, 0, stream>>>(
            _positions.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            _faces.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
            transform,
            _mesh_cdf.packed_accessor32<float, 1, torch::RestrictPtrTraits>());

        AT_CUDA_CHECK(hipGetLastError());
        AT_CUDA_CHECK(hipStreamSynchronize(stream));
    }

    {
        at::cuda::CUDAGuard device_guard {device};
        const auto stream = at::cuda::getCurrentCUDAStream();

        detail::computeMeshTriangleCDFKernel<<<1, 1, 0, stream>>>(
            _mesh_cdf.packed_accessor32<float, 1, torch::RestrictPtrTraits>());

        AT_CUDA_CHECK(hipGetLastError());
        AT_CUDA_CHECK(hipStreamSynchronize(stream));
    }

    data.total_area = _mesh_cdf.index({_mesh_cdf.size(0) - 1}).cpu().item<float>();
    {
        at::cuda::CUDAGuard device_guard {device};
        const auto stream = at::cuda::getCurrentCUDAStream();

        const int threads_per_block = 128;
        dim3 grid;
        at::cuda::getApplyGrid(_mesh_cdf.size(0), grid, device.index(), threads_per_block);
        dim3 threads = at::cuda::getApplyBlock(threads_per_block);

        detail::normalizeMeshTriangleCDFKernel<<<grid, threads, 0, stream>>>(
            _mesh_cdf.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
            data.total_area);

        AT_CUDA_CHECK(hipGetLastError());
        AT_CUDA_CHECK(hipStreamSynchronize(stream));
    }

    data.mesh_cdf       = (float*)_mesh_cdf.data_ptr();
    data.local_to_world = transform;
    data.world_to_local = glm::inverse(transform);
    data.num_faces      = _faces.size(0);

    _mesh_emitter_data.upload(&data);
}

MeshEmitter::~MeshEmitter()
{
    MeshEmitterData data;

    _mesh_emitter_data.download(&data);

    CUDA_SAFE_CALL(hipDestroyTextureObject(data.emissive_texture));

    CUDA_SAFE_CALL(hipFreeArray(_emissive_texture));
}

void MeshEmitter::initializeEmitter(const atcg::ref_ptr<RayTracingPipeline>& pipeline,
                                    const atcg::ref_ptr<ShaderBindingTable>& sbt)
{
    const std::string ptx_emitter_filename = "./build/ptxmodules.dir/Debug/EmitterKernels.ptx";
    auto sample_prog_group =
        pipeline->addCallableShader({ptx_emitter_filename, "__direct_callable__sample_meshemitter"});
    auto eval_prog_group = pipeline->addCallableShader({ptx_emitter_filename, "__direct_callable__eval_meshemitter"});
    uint32_t sample_idx  = sbt->addCallableEntry(sample_prog_group, _mesh_emitter_data.get());
    uint32_t eval_idx    = sbt->addCallableEntry(eval_prog_group, _mesh_emitter_data.get());

    EmitterVPtrTable table;
    table.sampleCallIndex = sample_idx;
    table.evalCallIndex   = eval_idx;

    _vptr_table.upload(&table);
}

EnvironmentEmitter::EnvironmentEmitter(const atcg::ref_ptr<Texture2D>& texture)
{
    auto environment_texture = texture->getData(atcg::GPU);

    EnvironmentEmitterData data;

    ::detail::convertToTextureObject(environment_texture, _environment_texture, data.environment_texture);

    _environment_emitter_data.upload(&data);
}

EnvironmentEmitter::~EnvironmentEmitter()
{
    EnvironmentEmitterData data;

    _environment_emitter_data.download(&data);

    CUDA_SAFE_CALL(hipDestroyTextureObject(data.environment_texture));

    CUDA_SAFE_CALL(hipFreeArray(_environment_texture));
}

void EnvironmentEmitter::initializeEmitter(const atcg::ref_ptr<RayTracingPipeline>& pipeline,
                                           const atcg::ref_ptr<ShaderBindingTable>& sbt)
{
    const std::string ptx_emitter_filename = "./build/ptxmodules.dir/Debug/EmitterKernels.ptx";
    auto sample_prog_group =
        pipeline->addCallableShader({ptx_emitter_filename, "__direct_callable__sample_environmentemitter"});
    auto eval_prog_group =
        pipeline->addCallableShader({ptx_emitter_filename, "__direct_callable__eval_environmentemitter"});
    uint32_t sample_idx = sbt->addCallableEntry(sample_prog_group, _environment_emitter_data.get());
    uint32_t eval_idx   = sbt->addCallableEntry(eval_prog_group, _environment_emitter_data.get());

    EmitterVPtrTable table;
    table.sampleCallIndex = sample_idx;
    table.evalCallIndex   = eval_idx;

    _vptr_table.upload(&table);
}
}    // namespace atcg