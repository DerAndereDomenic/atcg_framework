#include "hip/hip_runtime.h"
#pragma cuda_source_property_format = PTX

#include <Core/CUDA.h>

// #include <Integrator/PathtracingData.cuh>

#include <Core/TraceParameters.h>
#include <Math/Random.h>
#include <Core/SurfaceInteraction.h>
#include <Core/Payload.h>
#include "RadiosityParams.h"

extern "C"
{
    __constant__ RadiosityParams params;
}

extern "C" __global__ void __raygen__rg()
{
    uint3 launch_idx = optixGetLaunchIndex();

    if(launch_idx.x >= params.n_faces || launch_idx.y >= params.n_faces) return;

    if(launch_idx.x >= launch_idx.y) return;

    glm::u32vec3 tri_i = params.shape->faces[launch_idx.x];
    glm::u32vec3 tri_j = params.shape->faces[launch_idx.y];

    glm::vec3 p_i1 = params.shape->positions[tri_i.x];
    glm::vec3 p_i2 = params.shape->positions[tri_i.y];
    glm::vec3 p_i3 = params.shape->positions[tri_i.z];

    glm::vec3 p_j1 = params.shape->positions[tri_j.x];
    glm::vec3 p_j2 = params.shape->positions[tri_j.y];
    glm::vec3 p_j3 = params.shape->positions[tri_j.z];

    glm::vec3 n_i = glm::cross((p_i2 - p_i1), (p_i3 - p_i1));
    float area_i  = 0.5f * glm::length(n_i);

    glm::vec3 n_j = glm::cross((p_j2 - p_j1), (p_j3 - p_j1));
    float area_j  = 0.5f * glm::length(n_j);

    n_i /= (2.0f * area_i);
    n_j /= (2.0f * area_j);

    glm::vec3 c_i = (p_i1 + p_i2 + p_i3) / 3.0f;
    glm::vec3 c_j = (p_j1 + p_j2 + p_j3) / 3.0f;

    glm::vec3 ray_direction = c_j - c_i;
    float ray_distance      = glm::length(ray_direction);
    ray_direction /= ray_distance;

    float cos_theta_i = glm::max(0.0f, glm::dot(n_i, ray_direction));
    float cos_theta_j = glm::max(0.0f, -glm::dot(n_j, ray_direction));

    float G = 0.0f;
    if(cos_theta_i > 0 && cos_theta_j > 0)
    {
        bool occluded = atcg::traceOcclusion(params.handle,
                                             c_i,
                                             ray_direction,
                                             1e-4,
                                             ray_distance - 1e-4f,
                                             params.occlusion_trace_params);

        if(!occluded)
        {
            G = cos_theta_i * cos_theta_j / (ray_distance * ray_distance * glm::pi<float>());
        }
    }

    params.form_factors[launch_idx.x + params.n_faces * launch_idx.y] = G * area_j;
    params.form_factors[launch_idx.y + params.n_faces * launch_idx.x] = G * area_i;
}


extern "C" __global__ void __miss__occlusion()
{
    setOcclusionPayload(false);
}